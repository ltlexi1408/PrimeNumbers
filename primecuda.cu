#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>

__global__ void primeSieve(int* prime, int max){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = 2 + index; i <= sqrtf(max); i += stride){
        if(prime[i]){
            for(int j = i * i; j <= max; j += i){
                prime[j] = 0;
            }
        }
    }
    __syncthreads();
}

int main(){
    int max = 1000000000;
    // calcualte size of memory needed
    size_t size = (max + 1) * sizeof(int);

    // assign mem on host
    int* prime = (int*)malloc(size);
    for (int i = 0; i <= max; i++) prime[i] = 1;
    prime[0] = prime[1] = 0;

    // assign + cpy mem to Gpu
    int* primed;
    hipMalloc(&primed, size);
    hipMemcpy(primed, prime, size, hipMemcpyHostToDevice);

    // execute gpu code
    int block = 256;
    int numBlock = (max + block -1) / block;

    printf("%d:%d\n", block, numBlock);
    primeSieve <<<numBlock,block>>> (primed, max);

    hipMemcpy(prime, primed, size, hipMemcpyDeviceToHost);

    for(int i = 0; i <= max; i++){
        if(prime[i]){
            printf("%d, ", i);
        }
    }

    hipFree(primed);
    free(prime);

    return 0;
}